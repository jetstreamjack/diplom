#include <stdlib.h>
#include <stdio.h>

#include <QCoreApplication>

#include <vector>
#include <memory>
#include "balancer.h"
#include "gpu_controller.h"
#include "connection_observer.h"
#include "connection_controller.h"
#include "server.h"

int main(int argc, char ** argv)
{
	QCoreApplication a(argc, argv);

	hipError_t error;
	error = hipGetLastError();
	if (error != hipSuccess)
  	{
  		qDebug() << hipGetErrorString(error) << '\n';
  		exit(1);
  	}

	auto gpuController = std::make_unique<controller::GpuController>(); 
	auto balancer = std::make_unique<balancer::Balancer>(std::move(gpuController));
	
	auto connectionController = std::make_shared<connection_controller::ConnectionController>(std::move(balancer));
	auto connectionObserver = std::make_shared<connection::ConnectionObserver>();

	auto server = std::make_unique<server::Server>(connectionController, connectionObserver);

	server->Run(std::string{""});

	return a.exec();
}
