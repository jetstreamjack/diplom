#include "hip/hip_runtime.h"

#include "balancer.h"
#include "gpu_controller.h"

#include <vector>
#include <exception>
#include <string>

#include <algorithm>
#include <chrono>
#include <iterator>
#include <random>
#include <unordered_map>

#include <QCoreApplication>

namespace balancer {

__global__ void ProcessTask(int numFunc, double *path, double *res,
                            controller::GpuController *gpuController) {
  gpuController->InvokeClasterFunction(numFunc, path, res);

}

Balancer::Balancer(std::unique_ptr<controller::GpuController> gpuController)
    : m_gpuController(std::move(gpuController)) {
  // if(!m_gpuController)
  // {
  //     throw std::exception("GpuController is null!");
  // }
  // HANDLE_ERROR(hipStreamCreate(&stream0));

  hipMalloc((void **)&c_gpuController, sizeof(controller::GpuController));
  //controller::GpuController *gpu = new controller::GpuController();
  hipMemcpy(c_gpuController, m_gpuController.get(), sizeof(controller::GpuController),
             hipMemcpyHostToDevice);
}

// add commentary
TaskId Balancer::AddTask(int numFunc, PathVec path) {
  TaskId taskId = GenerateTaskId();

  m_taskMap.insert(std::pair<TaskId, Task>(taskId, Task(path)));

  auto error = hipGetLastError();
  if (error != hipSuccess) {
    qDebug() << "WTF ????!";
    printf("113 %s\n", hipGetErrorString(error));
    exit(1);
  }

  hipDeviceSynchronize();

  error = hipGetLastError();
  if (error != hipSuccess) {
    printf("16 %s\n", hipGetErrorString(error));
    exit(1);
  }

  ProcessTask<<<1, 1>>>(numFunc, m_taskMap.find(taskId)->second.m_cudaMem,
                        m_taskMap.find(taskId)->second.m_resutlMem,
                        c_gpuController);

  return taskId;
}
double Balancer::GetTaskResult(TaskId taskId) {
  hipDeviceSynchronize();
  auto res = m_taskMap.find(taskId)->second.GetResult();
  m_taskMap.erase(taskId);

  return res;
}

TaskId Balancer::GenerateTaskId() {
  std::mt19937 gen(std::chrono::system_clock::now().time_since_epoch().count());
  std::uniform_int_distribution<int> distr(0, 1000000);
  // TODO(odnorob): add check on unique
  return distr(gen);
}

Task::Task(const Task &task) {
  this->m_vecSize = task.m_vecSize;
  hipMalloc(&m_cudaMem, m_vecSize * sizeof(double));
  hipMemcpy(m_cudaMem, task.m_cudaMem, m_vecSize * sizeof(double),
             hipMemcpyDeviceToDevice);
  hipMalloc(&m_resutlMem, sizeof(double));
  hipDeviceSynchronize();

  auto error = hipGetLastError();
  if (error != hipSuccess) {
    printf("1123 %s\n", hipGetErrorString(error));
    exit(1);
  }
}

Task &Task::operator=(const Task &task) {
  m_vecSize = task.m_vecSize;
  hipMalloc(&m_cudaMem, m_vecSize * sizeof(double));
  hipMemcpy(m_cudaMem, task.m_cudaMem, m_vecSize * sizeof(double),
             hipMemcpyDeviceToDevice);
  hipMalloc(&m_resutlMem, sizeof(double));
  hipDeviceSynchronize();

  auto error = hipGetLastError();
  if (error != hipSuccess) {
    printf("1123 %s\n", hipGetErrorString(error));
    exit(1);
  }
}

Task::Task(PathVec path) : m_vecSize(path.size()) {
  hipMalloc(&m_cudaMem, path.size() * sizeof(double));
  hipMemcpy(m_cudaMem, path.data(), path.size() * sizeof(double),
             hipMemcpyHostToDevice);
  hipMalloc(&m_resutlMem, sizeof(double));
  hipDeviceSynchronize();

  auto error = hipGetLastError();
  if (error != hipSuccess) {
    printf(" 17 %s\n", hipGetErrorString(error));
    exit(1);
  }
}

Task::~Task() {
  hipDeviceSynchronize();
  hipFree(m_cudaMem);
  hipFree(m_resutlMem);
  hipDeviceSynchronize();
}

double Task::GetResult() {
  hipDeviceSynchronize();
  auto error = hipGetLastError();
  if (error != hipSuccess) {
    printf(" 2  %s\n", hipGetErrorString(error));
    exit(1);
  }
  double *result = (double *)malloc(sizeof(double));
  hipMemcpy(result, m_resutlMem, sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  double res = *result;
  printf("res2:%f \n", *result);
  free(result);

  return res;
}

} // namespace balancer
