#include "hip/hip_runtime.h"

#include "balancer.h"
#include "gpu_controller.h"
#include "i_balancer.h"
#include "i_gpu_controller.h"

#include <vector>
// #include <cstdin.h>
#include <exception>
#include <string>

#include <algorithm>
#include <chrono>
#include <iterator>
#include <random>
#include <unordered_map>

namespace balancer {

__global__ void ProcessTask(int numFunc, double *path, double *res,
                            controller::GpuController *gpuController) {
  path = path;
  res = res;
  numFunc = numFunc;
  // gpuController->InvokeClasterFunction(numFunc, path, res);
  gpuController->Dude(numFunc, path, res);

  //*res = resW;
}

Balancer::Balancer(std::unique_ptr<controller::IGpuController> gpuController)
    : m_gpuController(std::move(gpuController)) {
  // if(!m_gpuController)
  // {
  //     throw std::exception("GpuController is null!");
  // }
  // HANDLE_ERROR(hipStreamCreate(&stream0));
}

// add commentary
TaskId Balancer::AddTask(int numFunc, PathVec path) {
  TaskId taskId = GenerateTaskId();

  m_taskMap.insert(std::pair<TaskId, Task>(taskId, Task(path)));

  auto error = hipGetLastError();
  if (error != hipSuccess) {
    printf("113 %s\n", hipGetErrorString(error));
    exit(1);
  }

  // todo(odnorob): in ctor
  hipMalloc((void **)&gpuController, sizeof(controller::GpuController));
  controller::GpuController *gpu = new controller::GpuController();
  hipMemcpy(gpuController, gpu, sizeof(controller::GpuController),
             hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  error = hipGetLastError();
  if (error != hipSuccess) {
    printf("16 %s\n", hipGetErrorString(error));
    exit(1);
  }

  ProcessTask<<<1, 1>>>(numFunc, m_taskMap.find(taskId)->second.m_cudaMem,
                        m_taskMap.find(taskId)->second.m_resutlMem,
                        gpuController);

  return taskId;
}
double Balancer::GetTaskResult(TaskId taskId) {
  hipDeviceSynchronize();
  auto res = m_taskMap.find(taskId)->second.GetResult();
  m_taskMap.erase(taskId);
  return res;
}

TaskId Balancer::GenerateTaskId() {
  std::mt19937 gen(std::chrono::system_clock::now().time_since_epoch().count());
  std::uniform_int_distribution<int> distr(0, 1000000);
  // TODO(odnorob): add check on unique
  return distr(gen);
}

Task::Task(const Task &task) {
  this->m_vecSize = task.m_vecSize;
  hipMalloc(&m_cudaMem, m_vecSize * sizeof(double));
  hipMemcpy(m_cudaMem, task.m_cudaMem, m_vecSize * sizeof(double),
             hipMemcpyDeviceToDevice);
  hipMalloc(&m_resutlMem, sizeof(double));
  hipDeviceSynchronize();

  auto error = hipGetLastError();
  if (error != hipSuccess) {
    printf("1123 %s\n", hipGetErrorString(error));
    exit(1);
  }
}

Task &Task::operator=(const Task &task) {
  m_vecSize = task.m_vecSize;
  hipMalloc(&m_cudaMem, m_vecSize * sizeof(double));
  hipMemcpy(m_cudaMem, task.m_cudaMem, m_vecSize * sizeof(double),
             hipMemcpyDeviceToDevice);
  hipMalloc(&m_resutlMem, sizeof(double));
  hipDeviceSynchronize();

  auto error = hipGetLastError();
  if (error != hipSuccess) {
    printf("1123 %s\n", hipGetErrorString(error));
    exit(1);
  }
}

Task::Task(PathVec path) : m_vecSize(path.size()) {
  hipMalloc(&m_cudaMem, path.size() * sizeof(double));
  hipMemcpy(m_cudaMem, path.data(), path.size() * sizeof(double),
             hipMemcpyHostToDevice);
  hipMalloc(&m_resutlMem, sizeof(double));
  hipDeviceSynchronize();

  auto error = hipGetLastError();
  if (error != hipSuccess) {
    printf("17 %s\n", hipGetErrorString(error));
    exit(1);
  }
}

Task::~Task() {
  hipDeviceSynchronize();
  hipFree(m_cudaMem);
  hipFree(m_resutlMem);
  hipDeviceSynchronize();
}

double Task::GetResult() {
  hipDeviceSynchronize();
  auto error = hipGetLastError();
  if (error != hipSuccess) {
    printf("2 %s\n", hipGetErrorString(error));
    exit(1);
  }
  double *result = (double *)malloc(sizeof(double));
  hipMemcpy(result, m_resutlMem, sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  double res = *result;
  printf("res2:%f", *result);
  free(result);
  return res;
}

} // namespace balancer
